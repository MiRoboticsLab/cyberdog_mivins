#include "hip/hip_runtime.h"
// Copyright (c) 2023-2023 Beijing Xiaomi Mobile Software Co., Ltd. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <iostream>
#include <mivins/direct/fast_detection_gpu.h>

#define M_PI_LOCAL 3.14159265358979323846

struct Corner
{
  int x;        ///< x-coordinate of corner in the image.
  int y;        ///< y-coordinate of corner in the image.
  int level;    ///< pyramid level of the corner.
  float score;  ///< shi-tomasi score of the corner.
  float angle;  ///< for gradient-features: dominant gradient angle.
};

__device__ float getAngleAtPixelUsingHistogramCuda(const unsigned char *src_img, int idx, int idy, int halfpatch_size, int width, int height)
{
    int n_bins = 36;
    double hist[36] = {0.0};

    // angleHistogram
    constexpr double pi2 = 2.0 * M_PI_LOCAL;
    for (int dy = -halfpatch_size; dy <= halfpatch_size; ++dy)
    {
        for (int dx = -halfpatch_size; dx <= halfpatch_size; ++dx)
        {
            // gradientAndMagnitudeAtPixel
            int index_x = idx + dx;
            int index_y = idy + dy; 
            if (index_y < 0 || index_y >= height - 1 || index_x < 0 || index_x >= width - 1)
                continue;
            int ddx = src_img[index_y * width + index_x + 1] - src_img[index_y * width + index_x - 1];
            int ddy = src_img[(index_y + 1) * width + index_x] - src_img[(index_y - 1) * width + index_x];
            float mag = sqrt((float)(ddx * ddx + ddy * ddy));
            float angle = atan2((float)ddy, (float)ddx); 

            size_t bin = round( n_bins * (angle + M_PI_LOCAL) / pi2);
            bin = (bin < n_bins) ? bin : 0u;
            hist[bin] += mag;
        }
    }

    // smoothOrientationHistogram
    double prev = hist[n_bins - 1], h0 = hist[0];
    for (int i = 0; i < n_bins; ++i)
    {
        double temp = hist[i];
        hist[i] = 0.25 * prev + 0.5 * hist[i] + 0.25 * ( (n_bins == i + 1 ) ? h0 : hist[i + 1]);
        prev = temp;
    }

    // dgetDominantAngle
    double max_angle = hist[0];
    int max_bin = 0;
    for (int i = 1; i < n_bins; i++)
    {
        if (hist[i] > max_angle)
        {
            max_angle = hist[i];
            max_bin = i;
        }
    }

    return max_bin * 2.0 * M_PI_LOCAL / n_bins;
}

__global__ void Gaussian3x3Kernel(const unsigned char *src_data, unsigned char *gauss_data, const int width, const int height)
{
    const int id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (id_x >= width - 1 || id_y >= height - 1 || id_x == 0 || id_y == 0)
        return ;

    const int index = id_x + id_y * width;
    const int t_index = index - width;
    const int b_index = index + width;

    unsigned char ret = (src_data[t_index - 1] + src_data[t_index + 1] + src_data[b_index - 1] + src_data[b_index + 1] +
                      ((src_data[t_index] + src_data[index - 1] + src_data[index + 1] + src_data[b_index]) << 1) +
                       (src_data[index] << 2)) >> 4;
    gauss_data[index] = ret;
}

__global__ void ScharrForScore(const unsigned char *gauss_data, float *score_data,
                               const float threshold, const int border, const int width, const int height)
{
    const int id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (id_x >= width - border || id_y >= height - border || id_x < border || id_y < border)
        return;

    const int index = id_x + id_y * width;
    const int t_index = index - width;
    const int b_index = index + width;

    short dx = (short)((gauss_data[t_index + 1] + gauss_data[b_index + 1] - gauss_data[t_index - 1] - gauss_data[b_index - 1]) * 3 +
                       (gauss_data[index + 1] - gauss_data[index - 1]) * 10);
    short dy = (short)((gauss_data[b_index - 1] + gauss_data[b_index + 1] - gauss_data[t_index - 1] - gauss_data[t_index + 1]) * 3 +
                       (gauss_data[b_index] - gauss_data[t_index]) * 10);

    float mag = sqrt((float)(dx * dx + dy * dy));
    float score = 0.0f;
    if (mag > threshold)
    {
        score = mag;
    }
    score_data[index] = score;
}

__global__ void NonmaxSuppression(const unsigned char *src_img, float *score_data, void *p_corner, unsigned char *occupancy, int *mlock,
                                  const int border, const int scale, const int level, const int cell_size, const int cell_cols,
                                  const int width, const int height)
{
    const int id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (id_x >= width - border || id_y >= height - border || id_x < border || id_y < border)
        return;

    Corner *p = (Corner *)p_corner;
    int k = (int)( floor(id_y * scale / (float)cell_size) * cell_cols + floor(id_x * scale / (float)cell_size) );
    if (occupancy[k] == 1)
        return;
    
    const int index = id_x + id_y * width;
    const int t_index = index - width;
    const int b_index = index + width;

    if((score_data[index] > 0.0f) 
    && (score_data[index] >= score_data[t_index - 1])
    && (score_data[index] >= score_data[t_index]) 
    && (score_data[index] > score_data[t_index + 1])
    && (score_data[index] >= score_data[index - 1])
    && (score_data[index] > score_data[index + 1])
    && (score_data[index] >= score_data[b_index - 1])
    && (score_data[index] > score_data[b_index])
    && (score_data[index] > score_data[b_index + 1]) )
    {
        bool blocked = true;
        while (blocked)
        {
            if (0 == atomicCAS(&mlock[k], 0, 1))
            {
                if (score_data[index] > p[k].score)
                {
                    p[k].x = id_x * scale;
                    p[k].y = id_y * scale;
                    p[k].level = level - 1;
                    p[k].score = score_data[index];
                    //p[k].angle = getAngleAtPixelUsingHistogramCuda(src_img, id_x, id_y, 4, width, height);
                }
                __threadfence();
                atomicExch(&mlock[k], 0);
                blocked = false;
            }
        }
    }
}

__global__ void GetAngleByCorner(const unsigned char *src_img, void *p_corner, 
                                 const int threshold, const int scale, const int corner_size, const int width, const int height)
{
    const int id_x = blockIdx.x * blockDim.x + threadIdx.x;

    if (id_x >= corner_size)
        return;

    Corner *p = (Corner *)p_corner;

    if (p[id_x].score > (float)threshold)
    {
        p[id_x].angle = getAngleAtPixelUsingHistogramCuda(src_img, (p[id_x].x / scale), (p[id_x].y / scale), 4, width, height);
    }
}

namespace mivins
{
    namespace feature_detector_utils
    {
        void EdgeletDetectorV2Gpu(
            const ImgPyramid &img_pyr,
            const int threshold,
            const int border,
            const size_t min_level,
            const size_t max_level,
            Corners &corners,
            OccupandyGrid2D& grid)
        {
            constexpr int level = 1;
            constexpr int scale = (1 << level);
            int cell_size = grid.cell_size;

            void *src_img = (void *)img_pyr[level].data;
            int width = img_pyr[level].cols;
            int height = img_pyr[level].rows;
            size_t u8_data_size = width * height * sizeof(unsigned char);
            size_t corner_size = corners.size() * sizeof(Corner);

            dim3 block1(32);
            dim3 grid1((corners.size() - 1) / block1.x + 1);
            dim3 block2(cell_size, cell_size);
            dim3 grid2((width - 1) / block2.x + 1, (height - 1) / block2.y + 1);

            unsigned char *src_data = NULL;
            unsigned char *gauss_data = NULL; 
            unsigned char *p_corner = NULL;
            unsigned char *p_occupancy = NULL;
            float *score_data = NULL;
            int32_t *mlock = NULL;

            hipMallocManaged(&src_data, u8_data_size, hipMemAttachHost);
            hipMalloc(&gauss_data, u8_data_size);

            hipMalloc(&score_data, width * height * sizeof(float));
            hipMalloc(&mlock, corners.size() * sizeof(int32_t));
            hipMallocManaged(&p_corner, corner_size, hipMemAttachHost);
            hipMallocManaged(&p_occupancy, grid.occupancy_.size() * sizeof(unsigned char), hipMemAttachHost);

            memcpy((void *)src_data, src_img, u8_data_size);
            hipStreamAttachMemAsync(NULL, src_data, 0, hipMemAttachGlobal);

            Gaussian3x3Kernel<<<grid2, block2>>>(src_data, gauss_data, width, height);

            ScharrForScore<<<grid2, block2>>>(gauss_data, score_data, threshold, border, width, height);

            memcpy((void *)p_corner, corners.data(), corner_size);
            hipStreamAttachMemAsync(NULL, p_corner, 0, hipMemAttachGlobal);

            for (int i = 0; i < grid.occupancy_.size(); i++)
            {
                if(grid.occupancy_.at(i))
                    p_occupancy[i] = 1;
                else
                    p_occupancy[i] = 0;
            }

            hipStreamAttachMemAsync(NULL, p_occupancy, 0, hipMemAttachGlobal);
            hipMemset(mlock, 0, corners.size() * sizeof(int32_t));

            NonmaxSuppression<<<grid2, block2>>>(src_data, score_data, (void *)p_corner, p_occupancy, mlock,
                                                 border, scale, level, grid.cell_size, grid.n_cols,
                                                 width, height);

            GetAngleByCorner<<<grid1, block1>>>(src_data, p_corner, threshold, scale, corners.size(), width, height);

            hipStreamAttachMemAsync(NULL, p_corner, 0, hipMemAttachHost);
            hipStreamSynchronize(NULL);
            memcpy(corners.data(), p_corner, corner_size);

            hipFree(p_occupancy);
            hipFree(mlock);
            hipFree(p_corner);
            hipFree(score_data);
            hipFree(gauss_data);
            hipFree(src_data);
        }
    } //feature_detector_utils
} //feature_detector_utils