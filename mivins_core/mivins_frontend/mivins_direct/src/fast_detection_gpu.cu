#include "hip/hip_runtime.h"
// Copyright (c) 2023-2023 Beijing Xiaomi Mobile Software Co., Ltd. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "fast/fast.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <iostream>

#include <mivins/direct/fast_detection_gpu.h>
//#include <mivins/direct/feature_detector_utilities.h>
//#include <mivins/direct/feature_detector_types.h>
//#include <mivins/direct/feature_detector.h>

__device__ int MinMaxScore(int *scores, const int threshold) 
{
    int score_max = 0;

    for (int i = 0; i < 16; i++)
    {
        if (scores[(i + 0) & 0xf] > threshold &&
            scores[(i + 1) & 0xf] > threshold &&
            scores[(i + 2) & 0xf] > threshold &&
            scores[(i + 3) & 0xf] > threshold &&
            scores[(i + 4) & 0xf] > threshold &&
            scores[(i + 5) & 0xf] > threshold &&
            scores[(i + 6) & 0xf] > threshold &&
            scores[(i + 7) & 0xf] > threshold &&
            scores[(i + 8) & 0xf] > threshold &&
            scores[(i + 9) & 0xf] > threshold)
        {
            int score_min_tmp_0 = min(scores[(i + 0) & 0xf], scores[(i + 1) & 0xf]);
            int score_min_tmp_1 = min(scores[(i + 2) & 0xf], scores[(i + 3) & 0xf]);
            int score_min_tmp_2 = min(scores[(i + 4) & 0xf], scores[(i + 5) & 0xf]);
            int score_min_tmp_3 = min(scores[(i + 6) & 0xf], scores[(i + 7) & 0xf]);
            int score_min_tmp_4 = min(scores[(i + 8) & 0xf], scores[(i + 9) & 0xf]);
            score_min_tmp_0 = min(score_min_tmp_0, score_min_tmp_1);
            score_min_tmp_2 = min(score_min_tmp_2, score_min_tmp_3);
            score_min_tmp_0 = min(score_min_tmp_0, score_min_tmp_2);
            score_min_tmp_0 = min(score_min_tmp_0, score_min_tmp_4);
            score_max = max(score_max, score_min_tmp_0);
        }
        else if (scores[(i + 0) & 0xf] < -threshold && 
                 scores[(i + 1) & 0xf] < -threshold && 
                 scores[(i + 2) & 0xf] < -threshold && 
                 scores[(i + 3) & 0xf] < -threshold && 
                 scores[(i + 4) & 0xf] < -threshold && 
                 scores[(i + 5) & 0xf] < -threshold && 
                 scores[(i + 6) & 0xf] < -threshold && 
                 scores[(i + 7) & 0xf] < -threshold && 
                 scores[(i + 8) & 0xf] < -threshold && 
                 scores[(i + 9) & 0xf] < -threshold )
        {
            int score_min_tmp_0 = max(scores[(i + 0) & 0xf], scores[(i + 1) & 0xf]);
            int score_min_tmp_1 = max(scores[(i + 2) & 0xf], scores[(i + 3) & 0xf]);
            int score_min_tmp_2 = max(scores[(i + 4) & 0xf], scores[(i + 5) & 0xf]);
            int score_min_tmp_3 = max(scores[(i + 6) & 0xf], scores[(i + 7) & 0xf]);
            int score_min_tmp_4 = max(scores[(i + 8) & 0xf], scores[(i + 9) & 0xf]);
            score_min_tmp_0 = max(score_min_tmp_0, score_min_tmp_1);
            score_min_tmp_2 = max(score_min_tmp_2, score_min_tmp_3);
            score_min_tmp_0 = max(score_min_tmp_0, score_min_tmp_2);
            score_min_tmp_0 = max(score_min_tmp_0, score_min_tmp_4);
            score_max = max(score_max, abs(score_min_tmp_0));
        }
    }
    return score_max;
}

__device__ int Comparator(uint8_t *input, const int *circle, const int threshold, int index) 
{
    uint8_t pixel = input[index];
    int scores[16] = {0};

    /// iterate over whole circle
    for (size_t i = 0; i < 16; i++) // 16 + 10  --circle_size=16 --consecutive_point_size=10
    {
        scores[i] = pixel - input[index + circle[i]]; // i % circle_size

    }

    return MinMaxScore(scores, threshold + 1) - 1;
}

__global__ void FastKernel(uint8_t *src, int32_t *dst, int32_t *mlock, 
                           const int width, const int height, const int threshold, 
                           const int border, const int scale, const int cell_size, 
                           const int grid_size_x, const int level)
{
    int block_size = blockDim.x;
    int local_size = blockDim.x + 2;
    extern __shared__ int local_score[]; // block_size * block_size

    // get 1d coordinates and cutout borders
    const int pixel[16] = {
        0 + width * 3,
        1 + width * 3,
        2 + width * 2,
        3 + width * 1,
        3 + width * 0,
        3 + width * -1,
        2 + width * -2,
        1 + width * -3,
        0 + width * -3,
        -1 + width * -3,
        -2 + width * -2,
        -3 + width * -1,
        -3 + width * 0,
        -3 + width * 1,
        -2 + width * 2,
        -1 + width * 3,
    };

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int block_id = floor((float)idy * scale / cell_size) * grid_size_x + floor((float)idx * scale / cell_size);

    int start_index = threadIdx.y * blockDim.x + threadIdx.x;
    for (int index_local = start_index; index_local < local_size * local_size; index_local += block_size * block_size)
    {
        int y = blockIdx.y * blockDim.y - 1 + index_local / local_size;
        int x = blockIdx.x * blockDim.x - 1 + index_local % local_size;
        int index_global = y * width + x;

        if (x < 3 || y < 3 || x >= (width - 3) || y >= (height - 3))
        {
            local_score[index_local] = 0;
            continue;
        }
        local_score[index_local] = Comparator(src, pixel, threshold, index_global);
    }

    __syncthreads();
    int score_center_index = (threadIdx.y + 1) * local_size + threadIdx.x + 1;
    int score_center = local_score[score_center_index];

    bool blocked = true;
    if (score_center > local_score[score_center_index - local_size - 1] &&
        score_center > local_score[score_center_index - local_size - 0] &&
        score_center > local_score[score_center_index - local_size + 1] &&
        score_center > local_score[score_center_index - 1] &&
        score_center > local_score[score_center_index + 1] &&
        score_center > local_score[score_center_index + local_size - 1] &&
        score_center > local_score[score_center_index + local_size - 0] &&
        score_center > local_score[score_center_index + local_size + 1])
    {
        while (blocked)
        {
            if (0 == atomicCAS(&mlock[block_id], 0, 1))
            {
                if (idx >= border && idy >= border && idx < (width - border) && idy < (height - border))
                {
                    if ((score_center > *(dst + 4 * block_id + 3)) || 
                        ((score_center == *(dst + 4 * block_id + 3)) && 
                         (level == *(dst + 4 * block_id + 2)) &&
                         ((idy * scale < *(dst + 4 * block_id + 1)) || 
                          ((idy * scale == *(dst + 4 * block_id + 1)) && 
                           (idx * scale < *(dst + 4 * block_id))))))
                    {
                        *(dst + 4 * block_id    ) = idx * scale;
                        *(dst + 4 * block_id + 1) = idy * scale;
                        *(dst + 4 * block_id + 2) = level;
                        *(dst + 4 * block_id + 3) = score_center;
                    }
                }
                __threadfence();
                atomicExch(&mlock[block_id], 0);
                blocked = false;
            }
        }
    }
}

namespace mivins
{
    namespace feature_detector_utils
    {
        void FastDetectorGpu(
            const ImgPyramid &img_pyr,
            const int threshold,
            const int border,
            const size_t min_level,
            const size_t max_level,
            Corners &corners,
            OccupandyGrid2D& grid)
        {
            int block_size = grid.cell_size;
            int cell_size = grid.cell_size;
            int grid_size_x = grid.n_cols;
            int grid_size_y = grid.n_rows;
            int grid_size   = grid_size_x * grid_size_y;

            dim3 blocks(block_size, block_size);
            dim3 grids(std::ceil((float)img_pyr[0].cols / block_size), std::ceil((float)img_pyr[0].cols / block_size));
            uint8_t *p_mem_src = NULL;
            int32_t *p_mem_dst = NULL;
            int32_t *p_mlock = NULL;
            hipMallocManaged(&p_mem_src, img_pyr[0].total());
            hipMallocManaged(&p_mem_dst, grid_size * 16);
            hipMallocManaged(&p_mlock, grid_size * 4);
            hipMemset(p_mem_dst, 0, grid_size * 16);
            hipMemset(p_mlock, 0, grid_size * 4);

            for (int level = min_level; level <= max_level; level++)
            {
                const int scale  = (1 << level);
                const int width  = img_pyr[level].cols;
                const int height = img_pyr[level].rows;

                // hipMemcpy(p_mem_src, img_pyr[level].data, img_pyr[level].total(), hipMemcpyHostToDevice);
                hipStreamAttachMemAsync(NULL, p_mem_src, 0, hipMemAttachHost);
                memcpy(p_mem_src, (void *)img_pyr[level].data, img_pyr[level].total());
                hipStreamAttachMemAsync(NULL, p_mem_src, 0, hipMemAttachGlobal);

                FastKernel<<<grids, blocks, (block_size + 2) * (block_size + 2) * 4>>>(
                    p_mem_src, p_mem_dst, p_mlock, width, height, 
                    threshold, border, scale, cell_size, grid_size_x, level);
                hipDeviceSynchronize();
            }

            hipStreamAttachMemAsync(NULL, p_mem_dst, 0, hipMemAttachHost);
            for (int i = 0; i < grid_size; i++)
            {
                if (grid.occupancy_.at(i))
                    continue;

                if (p_mem_dst[4 * i + 3] > 0)
                {
                    corners.at(i).x = p_mem_dst[4 * i + 0];
                    corners.at(i).y = p_mem_dst[4 * i + 1];
                    corners.at(i).level = p_mem_dst[4 * i + 2];
                    corners.at(i).score = p_mem_dst[4 * i + 3];
                }
            }
            hipFree(p_mem_src);
            hipFree(p_mem_dst);
            hipFree(p_mlock);
        }
    } //feature_detector_utils
} //mivins